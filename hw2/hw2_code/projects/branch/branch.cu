#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
//#include <test1_kernel.cu>
// function [1 2 3 4 5 6 7 8 9 10]

// BIG DEVICE FUNCTION 
__device__ float bigfunction1(){
return ( exp2f( cosf( exp10f( log1pf( expm1f( logf( tanf( sqrtf( expf( sinf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction2(){
return (sinf( logf( cosf( log1pf( exp2f( tanf( exp10f( expf( sqrtf( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction3(){
return (exp2f( exp10f( expf( logf( cosf( log1pf( sinf( sqrtf( expm1f( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction4(){
return (exp10f( expm1f( exp2f( sinf( tanf( cosf( expf( sqrtf( log1pf( logf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction5(){
return (sqrtf( log1pf( logf( cosf( expf( sinf( exp10f( expm1f( tanf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction6(){
return (sinf( expf( sqrtf( expm1f( exp10f( cosf( logf( log1pf( tanf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction7(){
return (logf( exp2f( exp10f( expm1f( expf( cosf( log1pf( sqrtf( sinf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction8(){
return (sinf( log1pf( exp2f( expf( logf( tanf( expm1f( sqrtf( exp10f( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction9(){
return (logf( tanf( sinf( exp2f( log1pf( cosf( exp10f( sqrtf( expm1f( expf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction10(){
return (expf( cosf( exp2f( exp10f( expm1f( sinf( log1pf( tanf( logf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction11(){
return (exp2f( logf( expf( sqrtf( sinf( expm1f( cosf( tanf( exp10f( log1pf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction12(){
return (logf( cosf( exp2f( expm1f( sinf( exp10f( expf( log1pf( tanf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction13(){
return (sqrtf( logf( exp10f( expm1f( exp2f( tanf( log1pf( sinf( expf( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction14(){
return (log1pf( tanf( sinf( expm1f( logf( exp10f( cosf( exp2f( sqrtf( expf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction15(){
return (sinf( tanf( logf( exp2f( cosf( expf( exp10f( sqrtf( log1pf( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction16(){
return (cosf( exp2f( expm1f( sqrtf( expf( exp10f( tanf( sinf( logf( log1pf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction17(){
return (cosf( tanf( log1pf( expf( exp10f( exp2f( sinf( sqrtf( logf( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction18(){
return (sqrtf( sinf( expm1f( tanf( log1pf( cosf( expf( exp2f( exp10f( logf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction19(){
return (cosf( tanf( exp10f( exp2f( expf( expm1f( sinf( logf( log1pf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction20(){
return (sinf( exp2f( logf( expf( sqrtf( tanf( exp10f( expm1f( log1pf( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction21(){
return (expf( log1pf( sinf( tanf( cosf( logf( sqrtf( expm1f( exp10f( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction22(){
return (expf( tanf( sqrtf( exp10f( exp2f( expm1f( sinf( logf( log1pf( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction23(){
return (exp10f( sinf( tanf( log1pf( cosf( sqrtf( exp2f( logf( expm1f( expf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction24(){
return (logf( exp2f( expf( cosf( sinf( expm1f( sqrtf( log1pf( tanf( exp10f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction25(){
return (expf( exp2f( expm1f( exp10f( log1pf( logf( tanf( sinf( cosf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction26(){
return (expf( cosf( sqrtf( tanf( exp10f( sinf( log1pf( logf( expm1f( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction27(){
return (
exp2f( expf( log1pf( expm1f( exp10f( cosf( logf( sqrtf( sinf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction28(){
return (sinf( expf( expm1f( tanf( sqrtf( exp2f( cosf( log1pf( logf( exp10f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction29(){
return (exp10f( expf( logf( expm1f( log1pf( sqrtf( sinf( cosf( tanf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction30(){
return (exp2f( cosf( expm1f( exp10f( sqrtf( log1pf( expf( sinf( tanf( logf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction31(){
return (sqrtf( expm1f( exp10f( expf( cosf( tanf( exp2f( sinf( log1pf( logf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction32(){
return (log1pf( expf( logf( sqrtf( exp2f( tanf( sinf( cosf( exp10f( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction33(){
return (exp2f( logf( sqrtf( expf( exp10f( tanf( cosf( log1pf( expm1f( sinf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction34(){
return (sqrtf( log1pf( exp2f( expm1f( sinf( cosf( logf( tanf( exp10f( expf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction35(){
return (log1pf( sqrtf( exp2f( logf( expm1f( cosf( sinf( tanf( expf( exp10f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction36(){
return (log1pf( exp2f( exp10f( sinf( tanf( sqrtf( logf( expf( expm1f( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction37(){
return (expm1f( expf( cosf( exp2f( tanf( log1pf( exp10f( logf( sqrtf( sinf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction38(){
return (exp2f( logf( expf( expm1f( exp10f( log1pf( tanf( cosf( sinf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction39(){
return (exp2f( log1pf( sqrtf( expf( logf( sinf( exp10f( cosf( expm1f( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction40(){
return (exp2f( log1pf( sqrtf( cosf( sinf( expm1f( tanf( logf( exp10f( expf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction41(){
return (exp2f( expm1f( sinf( cosf( tanf( logf( expf( sqrtf( exp10f( log1pf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction42(){
return (exp10f( sqrtf( sinf( cosf( log1pf( logf( expf( expm1f( tanf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction43(){
return (sinf( expf( exp2f( logf( tanf( log1pf( expm1f( sqrtf( exp10f( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction44(){
return (log1pf( sqrtf( tanf( exp2f( sinf( exp10f( expf( logf( expm1f( cosf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction45(){
return (log1pf( cosf( tanf( sinf( logf( exp10f( expm1f( expf( sqrtf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction46(){
return (tanf( log1pf( cosf( expf( logf( exp10f( expm1f( exp2f( sinf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction47(){
return (expf( sinf( cosf( exp2f( expm1f( log1pf( exp10f( sqrtf( logf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction48(){
return (expf( exp10f( sqrtf( logf( expm1f( sinf( exp2f( cosf( log1pf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction49(){
return (cosf( exp10f( logf( expm1f( expf( sqrtf( tanf( log1pf( sinf( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction50(){
return (log1pf( expm1f( exp10f( sinf( exp2f( expf( cosf( logf( sqrtf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction51(){
return (exp2f( sqrtf( logf( cosf( log1pf( expf( exp10f( expm1f( sinf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction52(){
return (tanf( expm1f( logf( sqrtf( expf( sinf( cosf( log1pf( exp10f( exp2f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction53(){
return (cosf( sinf( expf( expm1f( log1pf( exp2f( tanf( logf( exp10f( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction54(){
return (log1pf( sqrtf( expf( sinf( exp2f( cosf( logf( expm1f( tanf( exp10f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction55(){
return (sinf( sqrtf( cosf( exp2f( tanf( logf( log1pf( expf( exp10f( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction56(){
return (cosf( sinf( exp10f( sqrtf( expf( tanf( exp2f( log1pf( logf( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction57(){
return (cosf( logf( exp10f( sqrtf( expm1f( log1pf( tanf( exp2f( expf( sinf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction58(){
return (sqrtf( log1pf( exp10f( cosf( expm1f( sinf( exp2f( logf( expf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction59(){
return (log1pf( expm1f( tanf( logf( expf( exp10f( exp2f( sinf( cosf( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction60(){
return (expm1f( expf( cosf( log1pf( logf( exp10f( tanf( sinf( exp2f( sqrtf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction61(){
return (expm1f( expf( sqrtf( sinf( exp2f( logf( exp10f( cosf( log1pf( tanf( float(threadIdx.x))))))))))));
}
__device__ float bigfunction62(){
return (logf( exp2f( sinf( tanf( cosf( log1pf( sqrtf( expf( expm1f( exp10f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction63(){
return (sinf( tanf( exp10f( expf( cosf( logf( log1pf( exp2f( sqrtf( expm1f( float(threadIdx.x))))))))))));
}
__device__ float bigfunction64(){
return (cosf( sinf( expf( expm1f( exp2f( tanf( sqrtf( exp10f( logf( log1pf( float(threadIdx.x))))))))))));
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel template for flops test
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    if (threadIdx.x < 128) {
	result = bigfunctiona();
    } else {
	result = bigfunctionb();
    }

     g_odata[0] = result;
}

// handles 1 branches in the code
__global__ void
runOnesBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    if (threadIdx.x == 0) {
			result = bigfunction1();
    } else {
			result = bigfunction2();
    }
}


// handles 2 branches in the code
__global__ void
runTwoBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    switch(threadIdx.x) {
			case 0:
				result = bigfunction1();
			break;
    	case 1:
				result = bigfunction2();
    	break;

			default:
				result = bigfunction3();
		}

		g_odata[threadIdx.x] = result;
}

// handles 4 branches in the code
__global__ void
runFourBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    switch(threadIdx.x) {
			case 0:
				result = bigfunction1();
			break;

    	case 1:
				result = bigfunction2();
    	break;

    	case 2:
				result = bigfunction2();
    	break;

			default:
				result = bigfunction3();
		}

		g_odata[threadIdx.x] = result;

}


// handles 8 branches in the code
__global__ void
runEightBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    switch(threadIdx.x) {
			case 0:
				result = bigfunction1();
			break;

    	case 1:
				result = bigfunction2();
    	break;

    	case 2:
				result = bigfunction3();
    	break;

    	case 3:
				result = bigfunction4();
    	break;

    	case 4:
				result = bigfunction5();
    	break;

    	case 5:
				result = bigfunction6();
    	break;

			case 6:
				result = bigfunction7();
    	break;

			case 7:
				result = bigfunction8();
    	break;

			default:
				result = bigfunction9();
		}

		g_odata[threadIdx.x] = result;
}
// handles 16 branches in the code
__global__ void
runSixteenBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    switch(threadIdx.x) {
			case 0:
				result = bigfunction1();
			break;

    	case 1:
				result = bigfunction2();
    	break;

    	case 2:
				result = bigfunction3();
    	break;

    	case 3:
				result = bigfunction4();
    	break;

    	case 4:
				result = bigfunction5();
    	break;

    	case 5:
				result = bigfunction6();
    	break;

			case 6:
				result = bigfunction7();
    	break;

			case 7:
				result = bigfunction8();
    	break;

			case 8:
				result = bigfunction9();
			break;

    	case 9:
				result = bigfunction10();
    	break;

    	case 10:
				result = bigfunction11();
    	break;

    	case 11:
				result = bigfunction12();
    	break;

    	case 12:
				result = bigfunction13();
    	break;

    	case 13:
				result = bigfunction14();
    	break;

			case 14:
				result = bigfunction15();
    	break;

			case 15:
				result = bigfunction16();
    	break;

			default:
				result = bigfunction17();
		}

		g_odata[threadIdx.x] = result;
}
// handles 32 branches in the code
__global__ void
runThirtyTwoBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    if (threadIdx.x < 128) {
	result = bigfunctiona();
    } else {
	result = bigfunctionb();
    }

     g_odata[0] = result;
}
// Handles 64 branches of the code
__global__ void
runSixtyFourBranches( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    if (threadIdx.x < 128) {
	result = bigfunctiona();
    } else {
	result = bigfunctionb();
    }

     g_odata[0] = result;
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{

    CUT_DEVICE_INIT();

		int num_branches = atoi(argv[1]);

    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    // adjust number of threads here
    unsigned int num_threads = 256;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    float* h_idata = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float* d_idata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    // adjust thread block sizes here
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
		switch (num_branches) {
			case 1:
    		runOneBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 2:
    		runTwoBranchestest<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 4:
    		runFourBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 8:
    		runEightBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 16:
    		runSixteenBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 32:
    		runThirtyTwoBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			case 64:
    		runSixtyFourBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
			default:
    		runNoBranches<<< grid, threads, mem_size >>>( d_idata, d_odata);
			break;
		}

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads,
                                hipMemcpyDeviceToHost) );

    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));

    // cleanup memory
    free( h_idata);
    free( h_odata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
}

/*logf( sqrtf( log1pf( tanf( expm1f( exp2f( cosf( expf( sinf( exp10f( 
exp10f( expm1f( log1pf( logf( exp2f( expf( sqrtf( cosf( sinf( tanf( 
expm1f( expf( sinf( exp10f( exp2f( cosf( sqrtf( logf( tanf( log1pf( 
sqrtf( sinf( exp2f( tanf( logf( log1pf( expm1f( expf( exp10f( cosf( 
expf( tanf( sqrtf( logf( exp2f( exp10f( cosf( expm1f( log1pf( sinf( 
tanf( exp2f( sinf( logf( expm1f( exp10f( cosf( sqrtf( log1pf( expf( 
expm1f( expf( cosf( logf( exp10f( tanf( sinf( sqrtf( log1pf( exp2f( 
exp2f( cosf( exp10f( sinf( sqrtf( expf( logf( log1pf( expm1f( tanf( 
logf( sinf( tanf( expf( sqrtf( exp2f( cosf( log1pf( expm1f( exp10f( 
sqrtf( exp2f( expm1f( cosf( logf( log1pf( tanf( expf( exp10f( sinf( 
exp10f( sqrtf( tanf( cosf( expf( logf( exp2f( log1pf( sinf( expm1f( 
logf( tanf( sinf( sqrtf( log1pf( expm1f( exp2f( cosf( expf( exp10f( 
sinf( sqrtf( log1pf( tanf( logf( cosf( exp2f( expm1f( expf( exp10f( 
log1pf( tanf( sqrtf( expm1f( exp2f( exp10f( logf( sinf( cosf( expf( 
sqrtf( exp2f( expm1f( logf( sinf( cosf( exp10f( expf( tanf( log1pf( 
exp2f( expm1f( sqrtf( cosf( logf( expf( sinf( tanf( exp10f( log1pf( 
sinf( sqrtf( log1pf( expf( expm1f( logf( tanf( exp2f( cosf( exp10f( 
logf( sinf( tanf( expf( cosf( exp2f( log1pf( sqrtf( expm1f( exp10f( 
logf( cosf( sqrtf( log1pf( expm1f( expf( sinf( exp2f( exp10f( tanf( 
expm1f( exp2f( log1pf( exp10f( logf( cosf( tanf( sqrtf( sinf( expf( 
sinf( cosf( tanf( exp2f( sqrtf( expm1f( exp10f( logf( expf( log1pf( 
log1pf( sqrtf( cosf( tanf( sinf( exp10f( expf( expm1f( exp2f( logf( 
log1pf( logf( exp10f( expm1f( tanf( expf( sqrtf( sinf( cosf( exp2f( 
exp2f( tanf( expf( cosf( expm1f( logf( sinf( sqrtf( log1pf( exp10f( 
expf( sinf( sqrtf( log1pf( expm1f( exp10f( tanf( logf( exp2f( cosf( 
expm1f( log1pf( expf( cosf( sqrtf( exp10f( logf( exp2f( tanf( sinf( 
log1pf( expf( expm1f( exp2f( sqrtf( tanf( sinf( exp10f( cosf( logf( 
expf( sinf( tanf( log1pf( logf( expm1f( exp2f( sqrtf( cosf( exp10f( 
sqrtf( tanf( log1pf( expf( expm1f( logf( cosf( exp10f( sinf( exp2f( 
expf( sinf( expm1f( sqrtf( tanf( log1pf( logf( cosf( exp10f( exp2f( 
cosf( exp10f( logf( expm1f( log1pf( sinf( tanf( sqrtf( exp2f( expf( 
expf( cosf( logf( log1pf( sinf( sqrtf( expm1f( exp10f( tanf( exp2f( 
exp2f( tanf( logf( sqrtf( cosf( sinf( expf( expm1f( log1pf( exp10f( 
expf( sinf( log1pf( tanf( exp10f( expm1f( sqrtf( logf( exp2f( cosf( 
log1pf( exp10f( sinf( expf( exp2f( tanf( logf( expm1f( sqrtf( cosf( 
expm1f( exp10f( sqrtf( tanf( expf( log1pf( exp2f( logf( sinf( cosf( 
sinf( sqrtf( exp10f( cosf( exp2f( expm1f( expf( tanf( log1pf( logf( 
exp2f( cosf( sinf( tanf( logf( expf( exp10f( sqrtf( log1pf( expm1f( 
log1pf( cosf( exp10f( sqrtf( exp2f( expf( sinf( tanf( logf( expm1f( 
expm1f( sqrtf( log1pf( logf( exp10f( sinf( expf( cosf( exp2f( tanf( 
tanf( expf( exp2f( logf( sqrtf( exp10f( cosf( log1pf( expm1f( sinf( 
exp2f( sinf( tanf( expf( sqrtf( log1pf( exp10f( logf( expm1f( cosf( 
sqrtf( logf( sinf( cosf( exp10f( exp2f( tanf( expf( expm1f( log1pf( 
log1pf( tanf( expf( exp2f( sinf( logf( expm1f( sqrtf( exp10f( cosf( 
sinf( cosf( sqrtf( expf( logf( log1pf( tanf( exp10f( expm1f( exp2f( 
exp10f( sqrtf( logf( expf( expm1f( sinf( cosf( tanf( exp2f( log1pf( 
expm1f( expf( exp10f( sqrtf( cosf( exp2f( log1pf( tanf( logf( sinf( 
exp2f( sqrtf( cosf( tanf( expf( log1pf( exp10f( sinf( expm1f( logf( 
expm1f( exp2f( exp10f( cosf( tanf( sinf( sqrtf( logf( expf( log1pf( 
expm1f( exp10f( sqrtf( cosf( exp2f( log1pf( tanf( logf( sinf( expf( 
cosf( exp10f( logf( sinf( expf( expm1f( tanf( log1pf( exp2f( sqrtf( 
cosf( tanf( sinf( sqrtf( exp2f( logf( exp10f( expf( log1pf( expm1f( 
exp10f( sinf( expm1f( expf( logf( tanf( cosf( log1pf( sqrtf( exp2f( 
logf( tanf( log1pf( exp10f( expf( sinf( expm1f( sqrtf( cosf( exp2f( 
cosf( sqrtf( sinf( expm1f( expf( exp2f( log1pf( tanf( exp10f( logf( 
sinf( expf( exp10f( log1pf( exp2f( logf( tanf( expm1f( sqrtf( cosf( 
expf( logf( exp2f( sqrtf( sinf( cosf( expm1f( tanf( log1pf( exp10f( 
sqrtf( exp10f( exp2f( log1pf( sinf( cosf( logf( expf( tanf( expm1f( 
expf( sqrtf( cosf( expm1f( sinf( log1pf( exp2f( tanf( exp10f( logf( 
exp2f( log1pf( sinf( expm1f( sqrtf( logf( cosf( tanf( exp10f( expf( 
sqrtf( sinf( logf( expf( exp10f( tanf( log1pf( exp2f( expm1f( cosf( 
sinf( cosf( exp2f( expm1f( exp10f( expf( logf( log1pf( sqrtf( tanf( 
expf( sqrtf( exp10f( expm1f( tanf( sinf( logf( exp2f( log1pf( cosf( 
sinf( sqrtf( tanf( expm1f( expf( cosf( log1pf( exp10f( exp2f( logf(*/ 

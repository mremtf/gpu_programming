#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
//#include <test1_kernel.cu>

__device__ float bigfunctiona()
{
return (expf(sqrtf(exp2f(exp10f(expm1f(logf(log1pf(sinf(cosf(tanf(float(threadIdx.x))))))))))));
}
__device__ float bigfunctionb()
{
return (sqrtf(expf(exp2f(exp10f(expm1f(logf(log1pf(sinf(cosf(tanf(float(threadIdx.x))))))))))));
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel template for flops test
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel( float* g_idata, float* g_odata) 
{
    float result=1;
 
    // place variety of branch solutions here
    // make sure you use results, so compiler does not optomize out
    if (threadIdx.x < 128) {
	result = bigfunctiona();
    } else {
	result = bigfunctionb();
    }

     g_odata[0] = result;
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{

    CUT_DEVICE_INIT();

    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    // adjust number of threads here
    unsigned int num_threads = 256;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    float* h_idata = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float* d_idata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    // adjust thread block sizes here
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads,
                                hipMemcpyDeviceToHost) );

    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));

    // cleanup memory
    free( h_idata);
    free( h_odata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
}

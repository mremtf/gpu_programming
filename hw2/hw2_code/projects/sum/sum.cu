#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#define NUM_BANKS 16

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <sum_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                      const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    CUT_DEVICE_INIT();

    int num_elements = 2;
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", &num_elements);

    unsigned int timer;
    CUT_SAFE_CALL( cutCreateTimer(&timer));
    
    const unsigned int num_threads = 2;
    const unsigned int mem_size = sizeof( float) * num_elements;

    float* h_data = (float*) malloc( mem_size);
      
    printf("INPUT: ");
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
        printf(" %f ", h_data[i]);
    }
    printf("\n");

    float* reference = (float*) malloc( mem_size);  
    computeGold( reference, h_data, num_elements);

    float* d_idata;
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );

#ifndef __DEVICE_EMULATION__
    dim3  grid(1, 1, 1);  
#else
    dim3  grid(1, 1, 1); // only one run block in device emu mode or it will be too slow
#endif
    dim3  threads(num_threads*2, 1, 1);

    CUT_CHECK_ERROR("Kernel execution failed");

    printf("Running sum of %d elements\n", num_elements);
  
    unsigned int numIterations = 100;

    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        sum_kernel<<< grid, threads, 2 * mem_size >>>
            (d_odata, d_idata, num_elements);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);

    cutResetTimer(timer);

    CUT_CHECK_ERROR("Kernel execution failed");

    CUDA_SAFE_CALL(hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                                   hipMemcpyDeviceToHost));

    printf("OUTPUT: ");
    printf(" %f ", h_data[0]);
    printf("\n");
    printf("REFERENCE: ");
    printf(" %f ", reference[0]);
    printf("\n");

    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected soluion
    
    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = cutComparefe( reference, h_data, 1, epsilon);
    printf( "sum: Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");

    free( h_data);
    free( reference);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}

#include "hip/hip_runtime.h"
#ifndef LOAD_LEVEL
#define LOAD_LEVEL 1
#endif

#ifndef BLOCK_LOAD_LEVEL
#define BLOCK_LOAD_LEVEL 1
#endif

#define XSTR(x) STR(x)
#define STR(x) #x

#pragma message "LOAD LEVEL = " XSTR(LOAD_LEVEL)
#pragma message "BLOCK LOAD LEVEL = " XSTR(BLOCK_LOAD_LEVEL)

#ifdef _WIN32
#define NOMINMAX
#endif

#define NUM_BANKS 16

// includes, system
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <sum_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

// regression test functionality
extern "C" unsigned int compare(const float *reference, const float *data, const unsigned int len);
extern "C" void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    runTest(argc, argv);
    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
    CUT_DEVICE_INIT();

    int num_elements = 2;
    cutGetCmdLineArgumenti(argc, (const char **) argv, "n", &num_elements);

    unsigned int timer;
    CUT_SAFE_CALL(cutCreateTimer(&timer));

    // Nope, 32. Always and forever.
    // Well, not always, but it's determined by the load level
    const unsigned int num_threads = 32 >> (LOAD_LEVEL - 1);
    if (num_threads == 0) {
        // num_threads is not a power of two, which is going to cause... issues. Die.
        printf("Load level too high (%d)! Exiting.", LOAD_LEVEL);
        exit(1);
    }

    // lazy re-adjust because we need it to be a multiple of 32 or we'll explode
    int correct_size = num_elements;
    if (correct_size % 32) {
        correct_size = (correct_size / 32 + 1) * 32;
    }
    unsigned int block_count = (correct_size / 32) >> (BLOCK_LOAD_LEVEL - 1);

    if (block_count == 0) {
        puts("Block count 0! Correcting to 1\n");
        block_count = 1;
    }

    const unsigned int mem_size = sizeof(float) * correct_size;

    float *h_data = (float *) malloc(mem_size);

    // printf("INPUT: ");
    for (unsigned int i = 0; i < num_elements; ++i) {
        h_data[i] = floorf(10 * (rand() / (float) RAND_MAX));
        // printf(" %f ", h_data[i]);
    }
    // blank out extras
    for (unsigned int i = num_elements; i < correct_size; ++i) {
        h_data[i] = 0.0f;
    }
    // printf("\n");

    float *reference = (float *) malloc(mem_size);
    computeGold(reference, h_data, num_elements);

    float *d_idata;
    float *d_odata;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_idata, mem_size));
    CUDA_SAFE_CALL(hipMalloc((void **) &d_odata, sizeof(float)));

    // sending a literal zero down because I don't want to memset it to 0x00 because
    // what if that is wrong for some reason.
    // AND I can't just have global id 0 blank it because it might get scheduled weird
    // It's dumb but deal with it.
    const float literal_zero = 0.0f;
    // CUDA_SAFE_CALL(hipMemcpy(d_odata, &literal_zero, sizeof(float), hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpy(d_idata, h_data, mem_size, hipMemcpyHostToDevice));

    CUT_CHECK_ERROR("Kernel execution failed");

    printf("Running sum of %d elements\n", num_elements);

    unsigned int numIterations = 100;

    printf("%d %d\n", block_count, num_threads, correct_size);

    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i) {
        CUDA_SAFE_CALL(hipMemcpy(d_odata, &literal_zero, sizeof(float), hipMemcpyHostToDevice));
        sum_kernel<<<block_count, num_threads, sizeof(float) * 32>>>(d_odata, d_idata, correct_size);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);

    cutResetTimer(timer);

    CUT_CHECK_ERROR("Kernel execution failed");

    CUDA_SAFE_CALL(hipMemcpy(h_data, d_odata, sizeof(float), hipMemcpyDeviceToHost));

    printf("OUTPUT: ");
    printf(" %f ", h_data[0]);
    printf("\n");
    printf("REFERENCE: ");
    printf(" %f ", reference[0]);
    printf("\n");

    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected soluion

    // We can use an epsilon of 0 since values are integral and in a range
    // that can be exactly represented
    float epsilon               = 0.0f;
    unsigned int result_regtest = cutComparefe(reference, h_data, 1, epsilon);
    printf("sum: Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");

    free(h_data);
    free(reference);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}

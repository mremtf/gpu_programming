#include "hip/hip_runtime.h"


////////////////////////////////////////////////////////////////////////////////
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>


__global__ void
simple_copy_kernel( float* g_idata, float* g_odata) 
{
		// thread copy	
		size_t gtid = blockDim.x * blockIdx.x + threadIdx.x;
		//printf("%lu ", gtid);
		g_odata[gtid] = g_idata[gtid] + 1;
}



void run_device_mem_local_to_gpu(float* h_idata, size_t h_size) {
    // adjust number of threads here
 		//unsigned int num_threads = h_size;
		// setup execution parameters
    // adjust thread block sizes here
		int grid_size = 0;
		int thread_count = 32;
		if ((h_size % thread_count) != 0) {
			grid_size = (h_size / thread_count + 1) * thread_count;
		}
		else {
			grid_size = h_size / thread_count;
		}


    unsigned int mem_size = sizeof( float) * h_size;

		unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));

    // allocate device memory
    float* d_idata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));


    CUT_SAFE_CALL( cutStartTimer( timer));
    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );
    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));

    // allocate device memory for result
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));
		
		
		printf ("blocks = %d\n", grid_size);
		
    dim3  grid( grid_size, 1, 1);
		
    dim3  threads( thread_count, 1, 1);

    // execute the selected kernel
    simple_copy_kernel<<< grid, threads, mem_size >>>( d_idata, d_odata);
		hipError_t cuerr = hipGetLastError() ;
		if( cuerr != hipSuccess) {
			printf("CUDA ERROR %d\n\n", cuerr);
		}
	
		// check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );


    // cleanup memory
    free( h_odata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));

}

void run_remote_peer_to_peer_memory_access(float* h_idata, size_t h_size) {
    // adjust number of threads here
 		//unsigned int num_threads = h_size;
		int grid_size = 0;
		int thread_count = 32;
		if ((h_size % thread_count) != 0) {
			grid_size = (h_size / thread_count + 1) * thread_count;
		}
		else {
			grid_size = h_size / thread_count;
		}
    unsigned int mem_size = sizeof( float) * h_size;

		hipSetDevice(0);
		// allocate device memory
		/*
		* DEVICE MEMORY ALLOCATIONS FOR DEVICE ONE
		*/
    
		float* d_idata, *d_odata;
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));

    // allocate device memory for result
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));
	
		/*
		* MEMORY COPIES FOR DEVICE ONE
		**/
		unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

		// copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

		CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Memcpy to card time: %f (ms)\n", cutGetTimerValue( timer));
		
    dim3  grid( grid_size, 1, 1);
		
    dim3  threads( thread_count, 1, 1);

    // execute the selected kernel
    simple_copy_kernel<<< grid, threads, mem_size >>>( d_idata, d_odata);

		// check if kernel execution generated and error
 		hipError_t cuerr = hipGetLastError(); 
		if( cuerr != hipSuccess) {
			printf("CUDA ERROR %d\n\n", cuerr);
		}
    CUT_CHECK_ERROR("Kernel execution failed");

		// change GPU device
		hipSetDevice(1);

		/*
		* DEVICE MEMORY ALLOCATIONS FOR DEVICE TWO
		**/

		// allocate device memory
    float* d_odata_two;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata_two, mem_size));

		// Allow access to data between cards
		hipDeviceEnablePeerAccess(0,0);
		// execute the selected kernel
		//CUDA_SAFE_CALL(hipMemcpyPeer(d_idata_two,1,d_idata,0,mem_size));

    // execute the selected kernel
    CUT_SAFE_CALL( cutStartTimer( timer));
    
		simple_copy_kernel<<< grid, threads, mem_size >>>( d_idata, d_odata_two);
		cuerr = hipGetLastError();
		if(  cuerr != hipSuccess) {
			printf("CUDA ERROR %d\n\n", cuerr);
		}

		// check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

		CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Memcpy to card time: %f (ms)\n", cutGetTimerValue( timer));
		/*
		* HOST MEMORY COPIES FROM DEVICE ONE
		**/

		hipSetDevice(0);
		// allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

		/*
		* HOST MEMORY COPIES FROM DEVICE TWO
		**/
		hipSetDevice(1);
		// allocate mem for the result on host side
    float* h_odata_two = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata_two, d_odata_two, mem_size,
                                hipMemcpyDeviceToHost) );

		if(memcmp(h_odata,h_odata_two,mem_size) != 0) {
			printf("FAILED TO BE EQUAL\n");
		} 

		/*
		* MEMORY CLEAN UP
		**/
	  // cleanup memory
    free( h_odata);
		free( h_odata_two);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUDA_SAFE_CALL(hipFree(d_odata_two));

    CUT_SAFE_CALL( cutDeleteTimer( timer));

}

void run_remote_memory_access_using_data_copy(float* h_idata, size_t h_size) {
    // adjust number of threads here
 		//unsigned int num_threads = h_size;
		int grid_size = 0;
		int thread_count = 32;
		if ((h_size % thread_count) != 0) {
			grid_size = (h_size / thread_count + 1) * thread_count;
		}
		else {
			grid_size = h_size / thread_count;
		}
    
		unsigned int mem_size = sizeof( float) * h_size;
//printf("%d!!!!!!!!\n",mem_size);
		// copy to GPU 
		
		// copy back from GPU
		// copy to another GPU



		hipSetDevice(0);
		// allocate device memory
    float* d_idata, *d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
	puts("FIRST ALLOCATION");

		unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
		CUT_SAFE_CALL( cutStartTimer( timer));

    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );
puts("MEMCPY");
		CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Memcpy to card time: %f (ms)\n", cutGetTimerValue( timer));

		 // allocate device memory for result
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));
		//printf ("blocks = %d\n", grid_size);
puts("SECOND ALLOCATION");		
    dim3  grid( grid_size, 1, 1);
		
    dim3  threads( thread_count, 1, 1);

    // execute the selected kernel
    simple_copy_kernel<<< grid, threads, mem_size >>>( d_idata, d_odata);
		hipError_t cuerr = hipGetLastError();
		if(  cuerr != hipSuccess) {
			printf("CUDA ERROR %d\n\n", cuerr);
			printf("ERROR: %s\n\n",hipGetErrorString(cuerr));
		}
		puts("FIRST RUN DONE");
		// check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
		// allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    
    // copy result from device to host
		CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

		// change GPU device
		hipSetDevice(1);
				// allocate mem for the result on host side
    float* h_odata_two = (float*) malloc( mem_size);
		// allocate device memory
    float* d_idata_two, *d_odata_two;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata_two, mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata_two, mem_size));

    CUT_SAFE_CALL( cutStartTimer( timer));

		CUDA_SAFE_CALL( hipMemcpy( d_idata_two, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

		CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Memcpy to card time: %f (ms)\n", cutGetTimerValue( timer));

    simple_copy_kernel<<< grid, threads, mem_size >>>( d_idata_two, d_odata_two);
		cuerr = hipGetLastError();
		if( cuerr != hipSuccess) {
			printf("CUDA ERROR %d\n\n", cuerr);
		}

		// check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata_two, d_odata_two, mem_size,
                                hipMemcpyDeviceToHost) );

		if(memcmp(h_odata,h_odata_two,mem_size) != 0) {
			printf("FAILED TO BE EQUAL\n");
		} 

	  // cleanup memory
    free( h_odata);
		free( h_odata_two);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_idata_two));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUDA_SAFE_CALL(hipFree(d_odata_two));
    CUT_SAFE_CALL( cutDeleteTimer( timer));

}

// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
		if (argc != 3) {
			printf("%s <memory access 1 local 2 peer-to-peer 3 peer-to-peer-memcpy> <num elements\n", argv[0]);
			return 0;
		}

    CUT_DEVICE_INIT();
		
		int memory_access = atoi(argv[1]);
		int num_elements = atoi(argv[2]);

    // allocate host memory
		if ((num_elements % 32) != 0) {
			num_elements = (num_elements / 32 + 1) * 32; 
		}
		printf("number_elements: %d\n\n", num_elements);

    float* h_idata = (float*) malloc( sizeof(float)* num_elements);
    // initalize the memory
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_idata[i] = (float) i;
    }

		switch (memory_access) {
			case 1:
				run_device_mem_local_to_gpu(h_idata,num_elements);
			break;
			case 2:	
				run_remote_peer_to_peer_memory_access(h_idata,num_elements);
			break;

			case 3:	
				run_remote_memory_access_using_data_copy(h_idata,num_elements);
			break;
			
			default:
				return -1;	
		}

		free(h_idata);

		return 0;
}

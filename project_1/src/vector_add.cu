#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>
#include <cstdint>
#include <functional>

#include <hip/hip_runtime_api.h>

#include "vector_add.hpp"
#include "parameters.hpp"
#include "device_queries.hpp"
#include "shared_utilities.hpp"
#include "timer.hpp"

using std::vector;

vector<float> cpu_addition(const vector<float> &a, const vector<float> &b) {
    vector<float> results(a);
    std::transform(a.begin(), a.end(), b.cbegin(), results.begin(), std::plus<float>());
    return results;
}

/*
calculate step, calculate final index, if step doesn't work, specify final index and special step

if special step doesn't exist, FFFFFFFF it. only warp divergence will be in one warp in final block.

ideally, last block on last device
*/

__global__ void cuda_vector_add(float *a, float *b, unsigned step, unsigned total, unsigned fix_position,
                                unsigned fix_step) {
    unsigned position = blockDim.x * blockIdx.x + threadIdx.x;
    position *= step;
    // printf("%d\t%d\t%d\t%d\n", blockDim.x, blockIdx.x, threadIdx.x, position);
    // This is a really dumb edge case clearly used to break the code, but
    // hell if I'm missing points for not catching when you request more threads than elements!
    if (position < total) {
        // Interesting thing to test
        // ternary here vs if. Only divergence should be last warp in last block
        // But the ternary will probably slow down everything?
        // It would avoid a warp divergence, though!
        if (position == fix_position) {
            step = fix_step;
        }
        a += position;
        b += position;
        for (int i = 0; i < step; ++i, ++a, ++b) {
            // printf("%p %p %i %i %f %f\n", a, b, position, i, *a, *b);
            *a += *b;
        }
    }
}

using device_config_t = struct {
    int device;
    void *vec_a_device, *vec_b_device;  // vecs gets summed into a
    unsigned step;
    unsigned fix_position;  // UINT_MAX
    unsigned fix_step;
};

void launch_kernels_and_report(const options_t &opts) {
    const int threads         = opts.threads;
    const int blocks          = opts.blocks;
    const bool validate       = opts.validate;
    const bool multi          = opts.multi;
    const double util         = opts.utilization;
    const size_t thread_total = blocks * threads;

    if (threads == 0 || blocks == 0) {
        throw std::runtime_error("Thread/Block count of 0!");
    }

    std::vector<int> devices = get_devices();
    if (!multi) {
        devices.resize(1);
    }
    const size_t num_devices = devices.size();

    std::vector<size_t> float_vec_size(num_devices);
    for (unsigned i = 0; i < num_devices; ++i) {
        float_vec_size[i] = get_global_mem(devices[i]) / sizeof(float) * util / 2.0;
        // number of total floats, get the utilization, div in two because a + b
        // resulting size is the size for vectors a and b
    }

    // Instead of making a giant contiguous vector and serving out slices to the devices
    // I'm just going to make smaller ones since there's no real difference

    std::vector<device_config_t> config(num_devices);
    for (unsigned i = 0; i < num_devices; ++i) {
        auto dim_pair = get_dims(devices[i]);
        if (dim_pair.first < threads || dim_pair.second < blocks) {
            throw std::runtime_error("Block/thread count outside device dims!");
        }
        config[i].device = devices[i];
        // config[i].a      = generate_vector(float_vec_size[i]);
        // config[i].b      = generate_vector(float_vec_size[i]);
        // config[i].c      = std::vector<float>(float_vec_size[i]);
        config[i].step = float_vec_size[i] / thread_total;
        if (config[i].step == 0) {
            std::cout << "More threads than values! Rude!" << std::endl;
            // with a very low mem utilization (read: testing)
            // it will end up with a step of 0 if you get total_threads over n_elem
            // So I guess hardcode 1 and nop anything off the end of the vector
            config[i].step         = 1;
            config[i].fix_position = UINT_MAX;
            config[i].fix_step     = 1;
        } else {
            const bool offset_needed = (config[i].step * thread_total) != float_vec_size[i];
            if (offset_needed) {
                config[i].fix_position = config[i].step * (thread_total - 1);
                config[i].fix_step     = config[i].step + (float_vec_size[i] - (config[i].step * thread_total));
            } else {
                config[i].fix_position = UINT_MAX;        // should never trigger
                config[i].fix_step     = config[i].step;  // but just in case
            }
        }
    }

    std::cout << "Configuration complete, generating data and executing." << std::endl;

    // prepare and launch! Woooooo.
    for (unsigned i = 0; i < num_devices; ++i) {
        timer time;
        /*
        std::cout << "Dev: " << config[i].device << " Step: " << config[i].step << " Fix_P: " << config[i].fix_position
                  << " Fix_s: " << config[i].fix_step << " Threads: " << thread_total
                  << " Val total: " << config[i].a.size() << std::endl;
        */

        std::vector<float> a = generate_vector(float_vec_size[i]);
        std::vector<float> b = generate_vector(float_vec_size[i]);
        std::vector<float> c = std::vector<float>(float_vec_size[i]);

        if (hipSetDevice(config[i].device) != hipSuccess) {
            throw std::runtime_error("could not select device!");
        }

        time.begin();

        if (hipMalloc(&config[i].vec_a_device, float_vec_size[i] * sizeof(float)) != hipSuccess
            || hipMalloc(&config[i].vec_b_device, float_vec_size[i] * sizeof(float)) != hipSuccess) {
            throw std::runtime_error("Failed to malloc vector!");
        }
        if (hipMemcpy(config[i].vec_a_device, a.data(), float_vec_size[i] * sizeof(float),
                       hipMemcpyHostToDevice)
                != hipSuccess
            || hipMemcpy(config[i].vec_b_device, b.data(), float_vec_size[i] * sizeof(float),
                          hipMemcpyHostToDevice)
                   != hipSuccess) {
            throw std::runtime_error("Failed to copy data to device!");
        }

        cuda_vector_add<<<blocks, threads>>>((float *) config[i].vec_a_device, (float *) config[i].vec_b_device,
                                             config[i].step, float_vec_size[i], config[i].fix_position,
                                             config[i].fix_step);

        if (hipMemcpy(c.data(), config[i].vec_a_device, float_vec_size[i] * sizeof(float),
                       hipMemcpyDeviceToHost)
            != hipSuccess) {
            throw std::runtime_error("Could not copy data back! (or kernel launch failed?)");
        }

        hipFree(config[i].vec_a_device);
        hipFree(config[i].vec_b_device);

        time.end();

        std::cout << "GPU_" << config[i].device << " time: " << time.ms_elapsed() << " ms" << std::endl;

        if (validate) {
            timer cpu_time;
            cpu_time.begin();
            std::vector<float> cpu_result = cpu_addition(a, b);
            cpu_time.end();
            std::cout << "CPU time: " << cpu_time.ms_elapsed() << " ms" << std::endl;
            if (!check_equal(c, cpu_result)) {
                std::cout << "VERIFICATION FAILED (epsilon issue?)" << std::endl;
                std::cout << a[0] << " " << a[1] << " " << a[2] << std::endl;
                std::cout << b[0] << " " << b[1] << " " << b[2] << std::endl << std::endl;

                std::cout << c[0] << " " << c[1] << " " << c[2] << std::endl;
                std::cout << cpu_result[0] << " " << cpu_result[1] << " " << cpu_result[2] << std::endl;
            }
        }
    }
}
